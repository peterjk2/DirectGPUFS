#include <hip/hip_runtime.h>
#include <memory>
#include <stdexcept>
#include <string>
#include <cstddef>
#include <cstdint>
#include <cstring>
#include "buffer.h"


/*
 * We need to use hipHostAlloc, since posix_memalign does not work well with CUDA.
 */
MemoryPtr allocateHostMemory(size_t size)
{
    void* pointer = nullptr;

    auto err = hipHostAlloc(&pointer, size, hipHostMallocDefault);
    if (err != hipSuccess)
    {
        throw std::runtime_error(std::string("Failed to allocate memory in local RAM: ") + hipGetErrorString(err));
    }

    memset(pointer, 0, size);
    return MemoryPtr(pointer, hipHostFree);
}

