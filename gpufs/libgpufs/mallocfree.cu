#include "hip/hip_runtime.h"
/* 
* This expermental software is provided AS IS. 
* Feel free to use/modify/distribute, 
* If used, please retain this disclaimer and cite 
* "GPUfs: Integrating a file system with GPUs", 
* M Silberstein,B Ford,I Keidar,E Witchel
* ASPLOS13, March 2013, Houston,USA
*/

/* 
* This expermental software is provided AS IS. 
* Feel free to use/modify/distribute, 
* If used, please retain this disclaimer and cite 
* "GPUfs: Integrating a file system with GPUs", 
* M Silberstein,B Ford,I Keidar,E Witchel
* ASPLOS13, March 2013, Houston,USA
*/


#ifndef MALLOCFREE_CU
#define MALLOCFREE_CU

#include "fs_constants.h"
#include "fs_debug.cu.h"
#include "fs_globals.cu.h"
#include "util.cu.h"
#include "mallocfree.cu.h"
#include "hashMap.cu.h"
#include <assert.h>


// MUST be called from a single thread
DEBUG_NOINLINE __device__  void PPool::init_thread(volatile Page* _storage) volatile
{
	rawStorage=_storage;

	int slice = PPOOL_FRAMES / NUM_MEMORY_RINGS;

	for( int i = 0; i < NUM_MEMORY_RINGS; ++i )
	{
		subRings[i].head = 0;
		subRings[i].tail = 0;
		subRings[i].swapLock = 0;
		subRings[i].base = i * slice;
		subRings[i].size = slice;
	}

	for(int i=0;i<PPOOL_FRAMES;i++)
	{
		frames[i].init_thread(&rawStorage[i],i);
		freeList[i]=i;
	}
}
	
DEBUG_NOINLINE __device__ volatile PFrame* PPool::allocPage() volatile
{
	PAGE_ALLOC_START_WARP

	int ringID = BLOCK_ID % NUM_MEMORY_RINGS;

	volatile uint	base = subRings[ringID].base;
	volatile uint&	swapLock = subRings[ringID].swapLock;
	volatile uint& 	head = subRings[ringID].head;
	volatile uint& 	tail = subRings[ringID].tail;
	volatile int& 	size = subRings[ringID].size;

	int oldSize = atomicSub( (int*) &size, 1 );

	if( LOWER_WATER_MARK < oldSize )
	{
		uint freeLoc = atomicInc( (uint*) &head, (PPOOL_FRAMES / NUM_MEMORY_RINGS) - 1 );
		volatile PFrame* pFrame = &( frames[freeList[base + freeLoc]] );

		GPU_ASSERT( freeList[base + freeLoc] == pFrame->rs_offset );

		PAGE_ALLOC_STOP_WARP

		return pFrame;
	}

	EVICT_START_WARP
	// else, we are almost out of memory
	if( MUTEX_TRY_LOCK(swapLock) )
	{
//		GDBGV("swapping", oldSize);
//		GPRINT("%d\n", oldSize);

		// swap
		uint numSwapped = 0;
		int numRetries = 0;
		uint candLoc = tail;

//		GDBGV("candLoc", candLoc);
		while( NUM_PAGES_SWAPOUT > numSwapped )
		{
			volatile PFrame* cand = &( frames[freeList[base + candLoc]] );

			// Try to remove from the hash
			bool removed = false;

			if( cand->dirty == 0 && cand->dirtyCounter == 0 )
			{
				removed = g_hashMap->removePFrame( cand );
			}

			if( removed )
			{
//				GDBGV("candLoc", candLoc);
				if( candLoc != tail )
				{
					// swap tail and current location
					uint t = freeList[base + tail];
					freeList[base + tail] = freeList[base + candLoc];
					freeList[base + candLoc] = t;

					__threadfence();
				}

				freePage( cand, tail, base );
				numSwapped++;
				candLoc = ( candLoc + 1 ) % (PPOOL_FRAMES / NUM_MEMORY_RINGS);
				continue;
			}

			// else
			// move it down the ring buffer since it's busy
//			uint moveLoc = ( candLoc + (PPOOL_FRAMES / 4) ) % PPOOL_FRAMES;
//			uint t = freeList[moveLoc];
//			freeList[moveLoc] = freeList[candLoc];
//			freeList[candLoc] = t;
//
//			threadfence();

			// Search for another one

			// In this case we will need to swap the element in tail to prevent loosing it later
			candLoc = ( candLoc + 1 ) % (PPOOL_FRAMES / NUM_MEMORY_RINGS);

			while( (NUM_SWAP_RETRIES > numRetries) || (0 == numSwapped) )
			{
				cand = &( frames[freeList[base + candLoc]] );

				bool removed = false;

				if( cand->dirty == 0 && cand->dirtyCounter == 0 )
				{
					removed = g_hashMap->removePFrame( cand );
				}

				if( removed )
				{
//					GDBGV("candLoc", candLoc);
					// swap tail and current location
					uint t = freeList[base + tail];
					freeList[base + tail] = freeList[base + candLoc];
					freeList[base + candLoc] = t;

					__threadfence();

					freePage( cand, tail, base );
					numSwapped++;
					candLoc = ( candLoc + 1 ) % (PPOOL_FRAMES / NUM_MEMORY_RINGS);
					break;
				}

				// move it down the ring buffer since it's busy
//				uint moveLoc = ( candLoc + (PPOOL_FRAMES / 4) ) % PPOOL_FRAMES;
//				uint t = freeList[moveLoc];
//				freeList[moveLoc] = freeList[candLoc];
//				freeList[candLoc] = t;
//
//				threadfence();

				candLoc = ( candLoc + 1 ) % (PPOOL_FRAMES / NUM_MEMORY_RINGS);
				numRetries++;

				GPU_ASSERT(numRetries < ((PPOOL_FRAMES / NUM_MEMORY_RINGS) / 2))
			}

			if( NUM_SWAP_RETRIES <= numRetries  )
			{
				break;
			}
		}

		GPU_ASSERT( numSwapped > 0 );

//		GDBGV("numSwapped", numSwapped);
//		GDBGV("numRetries", numRetries);

		uint freeLoc = atomicInc( (uint*) &head, (PPOOL_FRAMES / NUM_MEMORY_RINGS) - 1 );
		volatile PFrame* pFrame = &( frames[freeList[base + freeLoc]] );

		GPU_ASSERT( freeList[base + freeLoc] == pFrame->rs_offset );

		PAGE_ALLOC_STOP_WARP

		atomicAdd( (int*) &size, numSwapped );

		MUTEX_UNLOCK( swapLock );

		EVICT_STOP_WARP
		return pFrame;
	}
	else if( LOWER_WATER_MARK < oldSize )
	{
		uint freeLoc = atomicInc( (uint*) &head, PPOOL_FRAMES - 1 );
		volatile PFrame* pFrame = &( frames[freeList[base + freeLoc]] );

		GPU_ASSERT( freeList[base + freeLoc] == pFrame->rs_offset );

		PAGE_ALLOC_STOP_WARP
		EVICT_STOP_WARP

		return pFrame;
	}
	else
	{
		// Not enough memory, and someone is already swapping
		// Abort
		int old = atomicAdd( (int*) &size, 1 );
//		GDBGV("Revert malloc", old);
		EVICT_STOP_WARP
		return NULL;
	}
}

DEBUG_NOINLINE __device__ void PPool::freePage(volatile PFrame* frame, volatile unsigned int& tail, uint base) volatile {
	GPU_ASSERT( freeList[base + tail] == frame->rs_offset );

	frame->clean();
	freeList[base + tail] = frame->rs_offset;
	tail = ( tail + 1 ) % (PPOOL_FRAMES / NUM_MEMORY_RINGS);
	__threadfence();
}

//DEBUG_NOINLINE __device__ bool PPool::tryLockSwapper() volatile
//{
//	return MUTEX_TRY_LOCK(swapLock);
//}
//
//DEBUG_NOINLINE __device__ void PPool::lockSwapper() volatile
//{
//	MUTEX_LOCK(swapLock);
//}
//
//DEBUG_NOINLINE __device__ void PPool::unlockSwapper() volatile
//{
//	MUTEX_UNLOCK(swapLock);
//}

#endif
