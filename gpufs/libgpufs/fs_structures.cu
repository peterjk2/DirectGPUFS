#include "hip/hip_runtime.h"
/* 
 * This expermental software is provided AS IS.
 * Feel free to use/modify/distribute,
 * If used, please retain this disclaimer and cite
 * "GPUfs: Integrating a file system with GPUs",
 * M Silberstein,B Ford,I Keidar,E Witchel
 * ASPLOS13, March 2013, Houston,USA
 */

/* 
 * This expermental software is provided AS IS.
 * Feel free to use/modify/distribute,
 * If used, please retain this disclaimer and cite
 * "GPUfs: Integrating a file system with GPUs",
 * M Silberstein,B Ford,I Keidar,E Witchel
 * ASPLOS13, March 2013, Houston,USA
 */

#ifndef FS_STRUCTURES_CU
#define FS_STRUCTURES_CU

#include "fs_structures.cu.h"
#include "fs_globals.cu.h"

DEBUG_NOINLINE __device__ void PFrame::init_thread( volatile Page* _page, int _rs_offset ) volatile
{
	page = _page;
	rs_offset = _rs_offset;
	file_id = (uint) -1;
	content_size = 0;
	file_offset = (uint) -1;
	dirty = 0;
	dirtyCounter = 0;

	lock = 0;
	refCount = 0;
	state = INVALID;
}

DEBUG_NOINLINE __device__ void PFrame::clean() volatile
{
	file_id = (uint) -1;
	content_size = (uint)-1;
	file_offset = (uint) -1;
	dirty = 0;
	dirtyCounter = 0;
}

DEBUG_NOINLINE __device__ bool PFrame::try_lock_init(int ref) volatile
{
	if( MUTEX_WAS_LOCKED(lock) )
	{
		// page is busy
		// GDBG("try_lock_init busy", file_offset, refCount);
		return false;
	}

	if( INVALID == state )
	{
		// We are the ones initiating this page
		state = INIT;
		refCount = ref;
		__threadfence();
		// GDBG("try_lock_init", file_offset, refCount);
		// Keep lock
		return true;
	}
	else
	{
		// It is already initiated by someone else
		MUTEX_UNLOCK( lock );
		return false;
	}
}

DEBUG_NOINLINE __device__ void PFrame::unlock_init() volatile
{
	GPU_ASSERT( lock );
	GPU_ASSERT( state == INIT || state == UPDATING );

	state = VALID;
	MUTEX_UNLOCK( lock );
}



DEBUG_NOINLINE __device__ bool PFrame::try_lock_rw( int fd, int _version, size_t offset, int ref ) volatile
{
	if( MUTEX_WAS_LOCKED(lock) )
	{
		// page is busy
		// GDBG("try_lock_rw busy", offset, refCount);
		return false;
	}

	if( state == PFrame::VALID && fd == file_id && file_offset == offset )
	{
		if( version == _version )
		{
			// This is the right one
			refCount += ref;
			__threadfence();
			// GDBG("try_lock_rw", offset, refCount);
			MUTEX_UNLOCK( lock );
			return true;
		}
		else
		{
			// Right page but older version
			GPU_ASSERT( version < _version );
			GPU_ASSERT( refCount == 0 );

			state = UPDATING;
			refCount = ref;
			version = _version;
			__threadfence();
			// GDBG("weird", offset, refCount);
			// Keep lock
			return true;
		}
	}
	else
	{
		// page is either invalid or point to a different location
		MUTEX_UNLOCK( lock );
		return false;
	}
}

DEBUG_NOINLINE __device__ void PFrame::unlock_rw(int ref) volatile
{
	MUTEX_LOCK( lock );
	refCount -= ref;
	__threadfence();
	// GDBG("unlock_rw", file_offset, refCount);
	MUTEX_UNLOCK( lock );
}

DEBUG_NOINLINE __device__ void PFrame::lock_rw(int ref) volatile
{
	MUTEX_LOCK( lock );
	refCount += ref;
	MUTEX_UNLOCK( lock );
}

DEBUG_NOINLINE __device__ bool PFrame::try_invalidate( int fd, size_t offset ) volatile
{
	if( MUTEX_WAS_LOCKED(lock) )
	{
		// page is busy
		return false;
	}

	if( refCount == 0 && fd == file_id && offset == file_offset )
	{
		// We can safely remove this page
		clean();
		state = INVALID;
		__threadfence();
		MUTEX_UNLOCK( lock );
		return true;
	}
	else
	{
		// Someone is still using it, don't invalidate
		MUTEX_UNLOCK( lock );
		return false;
	}
}

DEBUG_NOINLINE __device__ void PFrame::markDirty() volatile
{
	dirty = 1;
}

DEBUG_NOINLINE __device__ void BusyList::init_thread() volatile
{
	for (int i = 0; i < NUM_BUSY_LISTS; ++i){
		_lock[i] = 0;
		heads[i] = NULL;
	}
}

DEBUG_NOINLINE __device__ void BusyList::clean() volatile
{
	for (int i = 0; i < NUM_BUSY_LISTS; ++i){
		heads[i] = NULL;
	}
}

DEBUG_NOINLINE __device__ void BusyList::push( volatile PFrame* frame ) volatile
{
	int id = BLOCK_ID % NUM_BUSY_LISTS;

	lock(id);

	volatile PFrame* temp = heads[id];
	heads[id] = frame;
	frame->nextDirty = temp;

	unlock(id);
}

DEBUG_NOINLINE __device__ void BusyList::lock(int id) volatile
{
	MUTEX_LOCK( _lock[id] );
}

DEBUG_NOINLINE __device__ bool BusyList::try_lock(int id) volatile
{
	return MUTEX_TRY_LOCK( _lock[id] );
}

DEBUG_NOINLINE __device__ void BusyList::unlock(int id) volatile
{
	MUTEX_UNLOCK( _lock[id] );
}

//******* OPEN/CLOSE *//

DEBUG_NOINLINE __device__ void FTable_entry::init_thread() volatile
{
	status = FSENTRY_EMPTY;
	refCount = 0;
	file_id = -1;
	cpu_fd = -1;
	version = 0;
	size = 0;
	drop_cache = 0;
	dirty = 0;
	cpu_timestamp = 0;
}

DEBUG_NOINLINE __device__ void FTable_entry::init( const volatile char* _filename, int _flags ) volatile
{
	strcpy_thread( filename, _filename, FILENAME_SIZE );
	status = FSENTRY_PENDING;
	refCount = 0;
	cpu_fd = -1;
	flags = _flags;
	did_open = 0;
	version++;
}

DEBUG_NOINLINE __device__ void FTable_entry::notify( int fd, int _cpu_fd, size_t _size,
		double timestamp, int _did_open ) volatile
{
	file_id = fd;
	cpu_fd = _cpu_fd;
	size = _size;
	cpu_timestamp = timestamp;
	did_open = _did_open;
	__threadfence();
	status = FSENTRY_OPEN;
	__threadfence();
}

DEBUG_NOINLINE __device__ void FTable_entry::wait_open() volatile
{
	WAIT_ON_MEM( status, FSENTRY_OPEN );
}

DEBUG_NOINLINE __device__ void FTable_entry::flush(bool closeFile) volatile
{
	if ( !dirty ) return;

	for (int id = 0; id < NUM_BUSY_LISTS; ++id) {
		BEGIN_SINGLE_THREAD
			busyList.lock(id);
		END_SINGLE_THREAD

		volatile PFrame* frame = busyList.heads[id];

		while( frame != NULL ) {
			__syncthreads();

			if( frame->dirty || frame->dirtyCounter>0 )	{
				__syncthreads();

				writeback_page_async_on_close(cpu_fd, frame, flags);
				frame->dirty = 0;
				frame->dirtyCounter = 0;
			}

			frame = frame->nextDirty;
		}

		BEGIN_SINGLE_THREAD
			busyList.unlock(id);
		END_SINGLE_THREAD
	}

	dirty = false;

	if (closeFile) {
		writeback_page_async_on_close_done(cpu_fd);
	}
}

DEBUG_NOINLINE __device__ void FTable_entry::clean() volatile
{
	GPU_ASSERT(refCount==0);
	status = FSENTRY_EMPTY;
	did_open = 0;
	dirty = 0;
}

DEBUG_NOINLINE __device__ void FTable_entry::close() volatile
{
	GPU_ASSERT(refCount==0);
	status = FSENTRY_CLOSED;
	did_open = 0;
	dirty = 0;
}

DEBUG_NOINLINE __device__ void FTable::lock() volatile
{
	MUTEX_LOCK( _lock );
}

DEBUG_NOINLINE __device__ void FTable::unlock() volatile
{
	MUTEX_UNLOCK( _lock );
}

DEBUG_NOINLINE __device__ void FTable::init_thread() volatile
{
	for( int i = 0; i < FSTABLE_SIZE; i++ )
	{
		files[i].init_thread();
		_lock = 0;
	}
}

DEBUG_NOINLINE __device__ int FTable::findEntry( volatile const char* filename, volatile bool* isNewEntry,
		int o_flags ) volatile
{
	*isNewEntry = true;
	int found = E_FSTABLE_FULL;
	int foundClosed = E_FSTABLE_FULL;

	for( int i = 0; i < FSTABLE_SIZE; i++ )
	{
		if( (found == E_FSTABLE_FULL) && (files[i].status == FSENTRY_EMPTY) )
		{
			found = i;
		}
		else if( !strcmp_thread( filename, files[i].filename, FILENAME_SIZE ) )
		{
			// found entry
			found = i;
			*isNewEntry = false;
			break;
		}
		else if( (foundClosed == E_FSTABLE_FULL) && (files[i].status == FSENTRY_CLOSED) )
		{
			foundClosed = i;
		}
	}

	if( *isNewEntry )
	{
		if( found != E_FSTABLE_FULL )
		{
			files[found].init( filename, o_flags );
			__threadfence();
			return found;
		}
		else if( foundClosed != E_FSTABLE_FULL )
		{
			files[foundClosed].init( filename, o_flags );
			__threadfence();
			return foundClosed;
		}
	}

	return found;
}
#endif
